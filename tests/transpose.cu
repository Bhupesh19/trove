#include "hip/hip_runtime.h"
#include <thrust/tuple.h>
#include <iostream>
#include <trove/utility.h>
#include <trove/transpose.h>
#include <trove/memory.h>
#include <thrust/device_vector.h>
#include "timer.h"

using namespace trove;


template<int size, typename T>
__global__ void test_transpose(T* r) {
    typedef array<T, size> Value;
    typedef array<int, size> Indices;
    int global_index = threadIdx.x + blockDim.x * blockIdx.x;

    Indices warp_offsets;
    int rotation;
    c2r_compute_indices(warp_offsets, rotation);

    Value data;
    data = counting_array<Value>::impl(
        global_index * size);
    
    for(int i = 0; i < 4096; i++) {
        c2r_warp_transpose(data, warp_offsets, rotation);
    }
    int warp_begin = threadIdx.x & (~WARP_MASK);
    int warp_idx = threadIdx.x & WARP_MASK;
    int warp_offset = (blockDim.x * blockIdx.x + warp_begin) * size;
    T* warp_ptr = r + warp_offset;
    warp_store(data, warp_ptr, warp_idx, 32);
}


template<int size, typename T>
__global__ void test_shared_transpose(T* r) {
    typedef array<T, size> Value;

    int global_index = threadIdx.x + blockDim.x * blockIdx.x;
    extern __shared__ T smem[];
    
    Value data;
    data = counting_array<Value>::impl(
        global_index * size);
    int warp_id = threadIdx.x >> 5;
    int warp_idx = threadIdx.x & WARP_MASK;

    for(int i = 0; i < 4096; i++) {
        volatile T* thread_ptr = smem + threadIdx.x * size;
        uncoalesced_store(data, thread_ptr);

        __syncthreads();
        data = warp_load<Value>(smem + warp_id * WARP_SIZE * size,
                                warp_idx);
        __syncthreads();
    }
    int warp_begin = threadIdx.x & (~WARP_MASK);
    int warp_offset = (blockDim.x * blockIdx.x + warp_begin) * size;
    T* warp_ptr = r + warp_offset;
    warp_store(data, warp_ptr, warp_idx, 32);
   
}

template<int size, typename T>
__global__ void test_unsafe_shared_transpose(T* r) {
    typedef array<T, size> Value;

    int global_index = threadIdx.x + blockDim.x * blockIdx.x;
    extern __shared__ T smem[];
    
    Value data;
    data = counting_array<Value>::impl(
        global_index * size);
    int warp_id = threadIdx.x >> 5;
    int warp_idx = threadIdx.x & WARP_MASK;

    for(int i = 0; i < 4096; i++) {
        volatile T* thread_ptr = smem + threadIdx.x * size;
        uncoalesced_store(data, thread_ptr);


        data = warp_load<Value>(smem + warp_id * WARP_SIZE * size,
                                warp_idx);
    }
    int warp_begin = threadIdx.x & (~WARP_MASK);
    int warp_offset = (blockDim.x * blockIdx.x + warp_begin) * size;
    T* warp_ptr = r + warp_offset;
    warp_store(data, warp_ptr, warp_idx, 32);
   
}



template<template<int> class F, typename Cons>
struct do_tests {
    static void impl() {
        F<Cons::head>::impl();
        do_tests<F, typename Cons::tail>::impl();
    }

};

template<template<int> class F>
struct do_tests<F, null_type> {
    static void impl() {}
};

template<int i>
struct run_shfl_transpose {
    static void impl() {
        
        int n_blocks = 13 * 8;
        int block_size = 256;

        thrust::device_vector<int> e(n_blocks*block_size*i);

        int max_iter = 10;
        cuda_timer t;
        t.start();
        for(int j = 0; j < max_iter; j++) {
            test_transpose<i><<<n_blocks, block_size>>>(
                thrust::raw_pointer_cast(e.data()));
        }
        float ms = t.stop();
        std::cout << "SHFL, " << i << ",  " << ms/max_iter << std::endl;
    }
};


template<int i>
struct run_unsafe_transpose {
    static void impl() {
        
        int n_blocks = 13 * 8;
        int block_size = 256;

        thrust::device_vector<int> e(n_blocks*block_size*i);

        cuda_timer t;
        t.start();
        int max_iter = 10;
        for(int j = 0; j < max_iter; j++) {
            test_unsafe_shared_transpose<i><<<n_blocks, block_size,
                sizeof(int) * i * block_size>>>(
                    thrust::raw_pointer_cast(e.data()));
        }
        float ms = t.stop();
        std::cout << "Unsafe, " << i << ",  " << ms/max_iter << std::endl;
    }
};

template<int i>
struct run_safe_transpose {
    static void impl() {
        
        int n_blocks = 13 * 8;
        int block_size = 256;

        thrust::device_vector<int> e(n_blocks*block_size*i);
        int max_iter = 10;
        cuda_timer t;
        t.start();
        for(int j = 0; j < max_iter; j++) {
            test_shared_transpose<i><<<n_blocks, block_size,
                sizeof(int) * i * block_size>>>(
                    thrust::raw_pointer_cast(e.data()));
        }
        float ms = t.stop();
        std::cout << "Safe, " << i << ",  " << ms/max_iter << std::endl;
    }
};

#ifndef LOWER_BOUND
#define LOWER_BOUND 1
#endif
#ifndef UPPER_BOUND
#define UPPER_BOUND 32
#endif

typedef static_range<LOWER_BOUND, UPPER_BOUND> sizes;


int main() {

    do_tests<run_shfl_transpose, sizes>::impl();
    do_tests<run_unsafe_transpose, sizes>::impl();
    do_tests<run_safe_transpose, sizes>::impl();
    
    
}
    
